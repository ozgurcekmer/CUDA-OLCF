#include "hip/hip_runtime.h"
#include "../include/GpuReduce.h"
#include <iostream>

// Matrix column-sum kernel
__global__
void reduce2Steps(const float* A, float* out, size_t ds)
{
	__shared__ float Tile[BLOCK_SIZE];
	size_t tID = threadIdx.x;
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	Tile[tID] = 0.0;
	const size_t STRIDE = gridDim.x * blockDim.x;

	while (idx < ds)
	{
		Tile[tID] = (Tile[tID] > A[idx]) ? Tile[tID] : A[idx];
		idx += STRIDE;
	}

	for (auto s = blockDim.x / 2; s > 0; s /= 2)
	{
		__syncthreads();
		if (tID < s)
		{
			Tile[tID] = (Tile[tID] > Tile[tID + s]) ? Tile[tID] : Tile[tID + s];
		}
	}

	if (tID == 0)
	{
		out[blockIdx.x] = Tile[0];
	}

}

template<typename T>
inline void GpuReduce<T>::deviceAllocations()
{
	gpuMalloc(&dA, SIZE_A);
	gpuMalloc(&dMax, SIZE_RED);
	gpuMalloc(&dMax0, SIZE_RED0);
	gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuReduce<T>::copyH2D()
{
	gpuMemcpy(this->dA, this->A.data(), SIZE_A, gpuMemcpyHostToDevice);
	gpuMemcpy(this->dMax, this->Max.data(), SIZE_RED, gpuMemcpyHostToDevice);
	gpuMemset(this->dMax0, 0, SIZE_RED0);
	gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuReduce<T>::copyD2H()
{
	gpuMemcpy(this->Max.data(), this->dMax, SIZE_RED, gpuMemcpyDeviceToHost);
	gpuCheckErrors("gpuMemcpy D2H failure");
}

template<typename T>
void GpuReduce<T>::launchSetup()
{
	auto blocksPerSM = 2048 / BLOCK_SIZE;
	int devID;
	int numSMs;
	gpuGetDevice(&devID);

	gpuDeviceGetAttribute(&numSMs, gpuDevAttrMultiProcessorCount, devID);
	std::cout << "There are " << numSMs << " SMs in this device." << std::endl;
	std::cout << "Blocks per SM: " << blocksPerSM << std::endl;

	gridSize = blocksPerSM * numSMs;
	std::cout << "Grid Size: " << gridSize << std::endl;
	std::cout << "Block Size: " << BLOCK_SIZE << std::endl;
}

template<typename T>
GpuReduce<T>::~GpuReduce()
{
	gpuFree(dA);
	gpuFree(dMax);
	gpuFree(dMax0);
	gpuCheckErrors("gpuFree failure");
}

template<typename T>
void GpuReduce<T>::solver()
{
	deviceAllocations();
	copyH2D();
	launchSetup();
	reduce2Steps << < gridSize, BLOCK_SIZE >> > ((float*)dA, (float*)dMax0, N);
	reduce2Steps << < 1, BLOCK_SIZE >> > ((float*)dMax0, (float*)dMax, gridSize);
	copyD2H();
}

template void GpuReduce<float>::deviceAllocations();
template void GpuReduce<double>::deviceAllocations();
template void GpuReduce<float>::copyH2D();
template void GpuReduce<double>::copyH2D();
template void GpuReduce<float>::copyD2H();
template void GpuReduce<double>::copyD2H();
template void GpuReduce<float>::solver();
template void GpuReduce<double>::solver();
template GpuReduce<float>::~GpuReduce();
template GpuReduce<double>::~GpuReduce();







