#include "hip/hip_runtime.h"
#include "../include/GpuSolver2.h"

#ifdef KERNELTIME
#include <omp.h>
#endif

using std::cout;
using std::endl;
using std::vector;

template <typename T>
__global__
void gpuSolver2(T* __restrict__ a, T* __restrict__ b)
{
    __shared__ T tile[TILE_DIM][TILE_DIM];

    int row = blockIdx.x * TILE_DIM + threadIdx.x;
    int col = blockIdx.y * TILE_DIM + threadIdx.y;
    //int width = gridDim.x * TILE_DIM;
    int width = N;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        tile[threadIdx.y + j][threadIdx.x] = a[INDX(col + j, row, N)];
    }

    __syncthreads();

    row = blockIdx.y * TILE_DIM + threadIdx.x;
    col = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        b[INDX(col + j, row, width)] = tile[threadIdx.x][threadIdx.y + j];
    }
    

}

template<typename T>
void GpuSolver2<T>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dA, BYTES);
    gpuMalloc(&dB, BYTES);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuSolver2<T>::copyH2D()
{
    gpuMemcpy(dA, this->A.data(), BYTES, gpuMemcpyHostToDevice);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuSolver2<T>::copyD2H()
{
    gpuMemcpy(this->B.data(), dB, BYTES, gpuMemcpyDeviceToHost);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template<typename T>
GpuSolver2<T>::~GpuSolver2()
{
    gpuFree(dA);
    gpuFree(dB);
    gpuCheckErrors("gpuFree failure");
}

template <typename T>
void GpuSolver2<T>::solver()
{

    deviceAllocations();

    copyH2D();
    dim3 threads(TILE_DIM, BLOCK_ROWS, 1);
    dim3 blocks(N / TILE_DIM, N / TILE_DIM, 1);
#ifdef KERNELTIME
    auto t0 = omp_get_wtime();
    gpuSolver2<T> << < blocks, threads >> > (dA, dB);
    gpuCheckErrors("gpu kernel launch failure");
    gpuDeviceSynchronize();
    auto t1 = omp_get_wtime();
    cout << "Kernel runtime: " << (t1 - t0) * 1000.0 << " ms." << endl;
#else
    gpuSolver2<T> << < blocks, threads >> > (dA, dB);
    gpuCheckErrors("gpu kernel launch failure");
#endif
    copyD2H();

}

template void GpuSolver2<float>::solver();
template void GpuSolver2<double>::solver();
template void GpuSolver2<float>::deviceAllocations();
template void GpuSolver2<double>::deviceAllocations();
template void GpuSolver2<float>::copyH2D();
template void GpuSolver2<double>::copyH2D();
template GpuSolver2<float>::~GpuSolver2();
template GpuSolver2<double>::~GpuSolver2();
