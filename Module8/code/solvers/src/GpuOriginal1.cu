#include "hip/hip_runtime.h"
#include "../include/GpuOriginal1.h"

#ifdef KERNELTIME
#include <omp.h>
#endif

using std::cout;
using std::endl;
using std::vector;

template <typename T>
__global__
void gpuOriginal1(T* __restrict__ a, T* __restrict__ b)
{
    int row = blockIdx.x * TILE_DIM + threadIdx.x;
    int col = blockIdx.y * TILE_DIM + threadIdx.y;
    //int width = gridDim.x * TILE_DIM;
    int width = N;

    if (row < N && col < N)
    {
        b[INDX(row, col, width)] = a[INDX(col, row, N)];
    }

}

template<typename T>
void GpuOriginal1<T>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dA, BYTES);
    gpuMalloc(&dB, BYTES);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuOriginal1<T>::copyH2D()
{
    gpuMemcpy(dA, this->A.data(), BYTES, gpuMemcpyHostToDevice);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuOriginal1<T>::copyD2H()
{
    gpuMemcpy(this->B.data(), dB, BYTES, gpuMemcpyDeviceToHost);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template<typename T>
GpuOriginal1<T>::~GpuOriginal1()
{
    gpuFree(dA);
    gpuFree(dB);
    gpuCheckErrors("gpuFree failure");
}

template <typename T>
void GpuOriginal1<T>::solver()
{

    deviceAllocations();

    copyH2D();
    dim3 threads(TILE_DIM, TILE_DIM, 1);
    dim3 blocks(N / TILE_DIM, N / TILE_DIM, 1);
#ifdef KERNELTIME
    auto t0 = omp_get_wtime();
    gpuOriginal1<T> << < blocks, threads >> > (dA, dB);
    gpuCheckErrors("gpu kernel launch failure");
    gpuDeviceSynchronize();
    auto t1 = omp_get_wtime();
    cout << "Kernel runtime: " << (t1 - t0) * 1000.0 << " ms." << endl;
#else
    gpuOriginal1<T> << < blocks, threads >> > (dA, dB);
    gpuCheckErrors("gpu kernel launch failure");
#endif
    copyD2H();

}

template void GpuOriginal1<float>::solver();
template void GpuOriginal1<double>::solver();
template void GpuOriginal1<float>::deviceAllocations();
template void GpuOriginal1<double>::deviceAllocations();
template void GpuOriginal1<float>::copyH2D();
template void GpuOriginal1<double>::copyH2D();
template GpuOriginal1<float>::~GpuOriginal1();
template GpuOriginal1<double>::~GpuOriginal1();
