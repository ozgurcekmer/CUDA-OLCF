#include "hip/hip_runtime.h"
#include "../include/GpuSolver3.h"

#ifdef KERNELTIME
#include <omp.h>
#endif

using std::cout;
using std::endl;
using std::vector;

template <typename T>
__global__
void gpuSolver3(T* __restrict__ a, T* __restrict__ b)
{
    __shared__ T tile[TILE_DIM][TILE_DIM + 1];

    int row = blockIdx.x * TILE_DIM + threadIdx.x;
    int col = blockIdx.y * TILE_DIM + threadIdx.y;
    //int width = gridDim.x * TILE_DIM;
    int width = N;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        tile[threadIdx.y + j][threadIdx.x] = a[INDX(col + j, row, N)];
    }

    __syncthreads();

    row = blockIdx.y * TILE_DIM + threadIdx.x;
    col = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        b[INDX(col + j, row, width)] = tile[threadIdx.x][threadIdx.y + j];
    }


}

template<typename T>
void GpuSolver3<T>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dA, BYTES);
    gpuMalloc(&dB, BYTES);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuSolver3<T>::copyH2D()
{
    gpuMemcpy(dA, this->A.data(), BYTES, gpuMemcpyHostToDevice);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuSolver3<T>::copyD2H()
{
    gpuMemcpy(this->B.data(), dB, BYTES, gpuMemcpyDeviceToHost);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template<typename T>
GpuSolver3<T>::~GpuSolver3()
{
    gpuFree(dA);
    gpuFree(dB);
    gpuCheckErrors("gpuFree failure");
}

template <typename T>
void GpuSolver3<T>::solver()
{

    deviceAllocations();

    copyH2D();
    dim3 threads(TILE_DIM, BLOCK_ROWS, 1);
    dim3 blocks(N / TILE_DIM, N / TILE_DIM, 1);
#ifdef KERNELTIME
    auto t0 = omp_get_wtime();
    gpuSolver3<T> << < blocks, threads >> > (dA, dB);
    gpuCheckErrors("gpu kernel launch failure");
    gpuDeviceSynchronize();
    auto t1 = omp_get_wtime();
    cout << "Kernel runtime: " << (t1 - t0) * 1000.0 << " ms." << endl;
#else
    gpuSolver3<T> << < blocks, threads >> > (dA, dB);
    gpuCheckErrors("gpu kernel launch failure");
#endif
    copyD2H();

}

template void GpuSolver3<float>::solver();
template void GpuSolver3<double>::solver();
template void GpuSolver3<float>::deviceAllocations();
template void GpuSolver3<double>::deviceAllocations();
template void GpuSolver3<float>::copyH2D();
template void GpuSolver3<double>::copyH2D();
template GpuSolver3<float>::~GpuSolver3();
template GpuSolver3<double>::~GpuSolver3();
